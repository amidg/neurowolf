#include <iostream>
#include <string>
#include <stdio.h>
#include <unistd.h>
#include "wolf.h"
#include "wolf.cpp"
#include <locale>
#include <hip/hip_runtime.h>

using namespace std;

/*
	GOAL:
	- design code to generate wolf wisdom using C++ and CUDA
	- C++ is used to generate text
	- CUDA is used to generate image/video content, can be used for text as well (not sure if needed)

*/

//sources for images, text files etc
string whereToFindSourceImages = "./ImgSource/";
string phrase1source = "./Source/Phrase1.txt";
string phrase2source = "./Source/Phrase2.txt";
string nounSource = "./Source/Noun.txt";
string verbSource = "./Source/Verb.txt";

//CUDA related code
hipDeviceProp_t deviceProp;
void CUDAdeviceInfo();

//wolf
Wolf newWolf = Wolf();

/*
	CUDA CODE IS EXECUTED USING THE FOLLOWING -> check NVIDIA documentation for this
	function<<<grid,block>>>(param1,param2,...); 
*/

int main() {
	//initializing code
    setlocale(LC_CTYPE,"Russian"); //russian language
	srand(time(NULL)); //initialize random seed

    //debug or temp code
    CUDAdeviceInfo();
	int i = 1;

    //loop code
	while(i < 10) {
		i++;
		newWolf.buildSimpleWisdomStructure(newWolf.getStringContentFromFile(phrase1source), 
										   newWolf.getStringContentFromFile(phrase2source) );

		cout << newWolf.getWisdom() << endl;
	}
	
	return 0;
}

//functions descriptions
void CUDAdeviceInfo() {
    printf("Wolf Wisdom Generator v10.2021.0\n");
    printf("Powered by NVIDIA CUDA 10.0 using:\n");

	hipGetDeviceProperties(&deviceProp, 0);

	printf("Device name:                %s\n", deviceProp.name); 
	printf("Major revision number:      %d\n", deviceProp.major);
    printf("Minor revision Number:      %d\n", deviceProp.minor); 
    printf("Total Global Memory:        %d\n", deviceProp.totalGlobalMem);
    printf("Total shared mem per block: %d\n", deviceProp.sharedMemPerBlock); 
    printf("Total const mem size:       %d\n", deviceProp.totalConstMem); 
    printf("Warp size:                  %d\n", deviceProp.warpSize); 
    printf("Maximum block dimensions:   %d x %d x %d\n", deviceProp.maxThreadsDim[0], \ 
														 deviceProp.maxThreadsDim[1], \
                                                         deviceProp.maxThreadsDim[2]); 

    printf("Maximum grid dimensions:    %d x %d x %d\n", deviceProp.maxGridSize[0], \
                                                         deviceProp.maxGridSize[1], \
                                                         deviceProp.maxGridSize[2]); 

    printf("Clock Rate:                 %d\n", deviceProp.clockRate); 
    printf("Number of muliprocessors:   %d\n", deviceProp.multiProcessorCount); 
}
