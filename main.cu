#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <stdio.h>
#include <unistd.h>
#include "wolf.h"
#include "wolf.cpp"
#include <locale>

using namespace std;

/*
	GOAL:
	- design code to generate wolf wisdom using C++ and CUDA
	- C++ is used to generate text
	- CUDA is used to generate image/video content, can be used for text as well (not sure if needed)

*/

//sources for images, text files etc
string whereToFindSourceImages = "./ImgSource/";
string phrase1source = "./Source/Phrase1.txt";
string phrase2source = "./Source/Phrase2.txt";
string nounSource = "./Source/Noun.txt";
string verbSource = "./Source/Verb.txt";

//function declarations
__global__ void cuda_hello();

//wolf
Wolf newWolf = Wolf();

/*
	CUDA CODE IS EXECUTED USING THE FOLLOWING -> check NVIDIA documentation for this
	function<<<grid,block>>>(param1,param2,...); 
*/

int main() {
	//initializing code
    setlocale(LC_CTYPE,"Russian"); //russian language
	srand (time(NULL)); //initialize random seed

    //debug or temp code
    cuda_hello<<<1,1>>>();
    std::cout << newWolf.getPhrase("Phrase1.txt");


    //loop code
	
	
	return 0;
}

//functions descriptions
__global__ void cuda_hello() {
    printf("Wolf Wisdom Generator v10.2021.0\n");
    printf("Powered by NVIDIA CUDA 10.0 using NVIDIA QUADRO K5000\n");
}
